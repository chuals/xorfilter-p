
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

#ifndef XOR_MAX_ITERATIONS
#define XOR_MAX_ITERATIONS 100 // probabillity of success should always be > 0.5 so 100 iterations is highly unlikely
#endif 

/**
 * We need a decent random number generator.
 **/

 // returns random number, modifies the seed
static inline uint64_t xor_rng_splitmix64(uint64_t* seed) {
    uint64_t z = (*seed += UINT64_C(0x9E3779B97F4A7C15));
    z = (z ^ (z >> 30)) * UINT64_C(0xBF58476D1CE4E5B9);
    z = (z ^ (z >> 27)) * UINT64_C(0x94D049BB133111EB);
    return z ^ (z >> 31);
}

/**
 * xor8 is the recommended default, no more than
 * a 0.3% false-positive probability.
 */
typedef struct xor8_s {
    uint64_t seed;
    uint64_t blockLength;
    uint8_t
        * fingerprints; // after xor8_allocate, will point to 3*blockLength values
} xor8_t;

struct xor_xorset_s {
    uint64_t xormask;
    uint32_t count;
};

typedef struct xor_xorset_s xor_xorset_t;

struct xor_keyindex_s {
    uint64_t hash;
    uint32_t index;
};

typedef struct xor_keyindex_s xor_keyindex_t;

struct xor_hashes_s {
    uint64_t h;
    uint32_t h0;
    uint32_t h1;
    uint32_t h2;
};

typedef struct xor_hashes_s xor_hashes_t;

static inline uint64_t xor_murmur64(uint64_t h) {
    h ^= h >> 33;
    h *= UINT64_C(0xff51afd7ed558ccd);
    h ^= h >> 33;
    h *= UINT64_C(0xc4ceb9fe1a85ec53);
    h ^= h >> 33;
    return h;
}

static inline uint64_t xor_mix_split(uint64_t key, uint64_t seed) {
    return xor_murmur64(key + seed);
}

static inline uint64_t xor_rotl64(uint64_t n, unsigned int c) {
    return (n << (c & 63)) | (n >> ((-c) & 63));
}

static inline uint32_t xor_reduce(uint32_t hash, uint32_t n) {
    // http://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
    return (uint32_t)(((uint64_t)hash * n) >> 32);
}

static inline uint32_t xor8_get_h0(uint64_t hash, const xor8_t* filter) {
    uint32_t r0 = (uint32_t)hash;
    return xor_reduce(r0, filter->blockLength);
}
static inline uint32_t xor8_get_h1(uint64_t hash, const xor8_t* filter) {
    uint32_t r1 = (uint32_t)xor_rotl64(hash, 21);
    return xor_reduce(r1, filter->blockLength);
}
static inline uint32_t xor8_get_h2(uint64_t hash, const xor8_t* filter) {
    uint32_t r2 = (uint32_t)xor_rotl64(hash, 42);
    return xor_reduce(r2, filter->blockLength);
}

static inline xor_hashes_t xor8_get_h0_h1_h2(uint64_t k, const xor8_t* filter) {
    uint64_t hash = xor_mix_split(k, filter->seed);
    xor_hashes_t answer;
    answer.h = hash;
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)xor_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)xor_rotl64(hash, 42);

    answer.h0 = xor_reduce(r0, filter->blockLength);
    answer.h1 = xor_reduce(r1, filter->blockLength);
    answer.h2 = xor_reduce(r2, filter->blockLength);
    return answer;
}

static inline uint64_t xor_fingerprint(uint64_t hash) {
    return hash ^ (hash >> 32);
}

// report memory usage
static inline size_t xor8_size_in_bytes(const xor8_t* filter) {
    return 3 * filter->blockLength * sizeof(uint8_t) + sizeof(xor8_t);
}

// release memory
static inline void xor8_free(xor8_t* filter) {
    free(filter->fingerprints);
    filter->fingerprints = NULL;
    filter->blockLength = 0;
}

// allocate enough capacity for a set containing up to 'size' elements
// caller is responsible to call xor8_free(filter)
static inline bool xor8_allocate(uint32_t size, xor8_t* filter) {
    size_t capacity = 32 + 1.23 * size;
    capacity = capacity / 3 * 3;
    filter->fingerprints = (uint8_t*)malloc(capacity * sizeof(uint8_t));
    if (filter->fingerprints != NULL) {
        filter->blockLength = capacity / 3;
        return true;
    }
    else {
        return false;
    }
}

//
// construct the filter, returns true on success, false on failure.
// most likely, a failure is due to too high a memory usage
// size is the number of keys
// The caller is responsable for calling xor8_allocate(size,filter) before.
// The caller is responsible to ensure that there are no duplicated keys.
// The inner loop will run up to XOR_MAX_ITERATIONS times (default on 100),
// it should never fail, except if there are duplicated keys. If it fails,
// a return value of false is provided.
//
bool xor8_populate(const uint64_t* keys, uint32_t size, xor8_t* filter) {
    uint64_t rng_counter = 1;
    filter->seed = xor_rng_splitmix64(&rng_counter);
    size_t arrayLength = filter->blockLength * 3; // size of the backing array
    size_t blockLength = filter->blockLength;

    xor_xorset_t* sets =
        (xor_xorset_t*)malloc(arrayLength * sizeof(xor_xorset_t));

    xor_keyindex_t* Q =
        (xor_keyindex_t*)malloc(arrayLength * sizeof(xor_keyindex_t));

    xor_keyindex_t* stack =
        (xor_keyindex_t*)malloc(size * sizeof(xor_keyindex_t));

    if ((sets == NULL) || (Q == NULL) || (stack == NULL)) {
        free(sets);
        free(Q);
        free(stack);
        return false;
    }
    xor_xorset_t* sets0 = sets;
    xor_xorset_t* sets1 = sets + blockLength;
    xor_xorset_t* sets2 = sets + 2 * blockLength;
    xor_keyindex_t* Q0 = Q;
    xor_keyindex_t* Q1 = Q + blockLength;
    xor_keyindex_t* Q2 = Q + 2 * blockLength;

    int iterations = 0;

    while (true) {
        iterations++;
        if (iterations > XOR_MAX_ITERATIONS) {
            fprintf(stderr, "Too many iterations. Are all your keys unique?");
            free(sets);
            free(Q);
            free(stack);
            return false;
        }

        memset(sets, 0, sizeof(xor_xorset_t) * arrayLength);
        for (size_t i = 0; i < size; i++) {
            uint64_t key = keys[i];
            xor_hashes_t hs = xor8_get_h0_h1_h2(key, filter);
            sets0[hs.h0].xormask ^= hs.h;
            sets0[hs.h0].count++;
            sets1[hs.h1].xormask ^= hs.h;
            sets1[hs.h1].count++;
            sets2[hs.h2].xormask ^= hs.h;
            sets2[hs.h2].count++;
        }
        // todo: the flush should be sync with the detection that follows
        // scan for values with a count of one
        size_t Q0size = 0, Q1size = 0, Q2size = 0;
        for (size_t i = 0; i < filter->blockLength; i++) {
            if (sets0[i].count == 1) {
                Q0[Q0size].index = i;
                Q0[Q0size].hash = sets0[i].xormask;
                Q0size++;
            }
        }

        for (size_t i = 0; i < filter->blockLength; i++) {
            if (sets1[i].count == 1) {
                Q1[Q1size].index = i;
                Q1[Q1size].hash = sets1[i].xormask;
                Q1size++;
            }
        }
        for (size_t i = 0; i < filter->blockLength; i++) {
            if (sets2[i].count == 1) {
                Q2[Q2size].index = i;
                Q2[Q2size].hash = sets2[i].xormask;
                Q2size++;
            }
        }

        size_t stack_size = 0;
        while (Q0size + Q1size + Q2size > 0) {
            while (Q0size > 0) {
                xor_keyindex_t keyindex = Q0[--Q0size];
                size_t index = keyindex.index;
                if (sets0[index].count == 0)
                    continue; // not actually possible after the initial scan.
                  //sets0[index].count = 0;
                uint64_t hash = keyindex.hash;
                uint32_t h1 = xor8_get_h1(hash, filter);
                uint32_t h2 = xor8_get_h2(hash, filter);

                stack[stack_size] = keyindex;
                stack_size++;
                sets1[h1].xormask ^= hash;
                sets1[h1].count--;
                if (sets1[h1].count == 1) {
                    Q1[Q1size].index = h1;
                    Q1[Q1size].hash = sets1[h1].xormask;
                    Q1size++;
                }
                sets2[h2].xormask ^= hash;
                sets2[h2].count--;
                if (sets2[h2].count == 1) {
                    Q2[Q2size].index = h2;
                    Q2[Q2size].hash = sets2[h2].xormask;
                    Q2size++;
                }
            }
            while (Q1size > 0) {
                xor_keyindex_t keyindex = Q1[--Q1size];
                size_t index = keyindex.index;
                if (sets1[index].count == 0)
                    continue;
                //sets1[index].count = 0;
                uint64_t hash = keyindex.hash;
                uint32_t h0 = xor8_get_h0(hash, filter);
                uint32_t h2 = xor8_get_h2(hash, filter);
                keyindex.index += blockLength;
                stack[stack_size] = keyindex;
                stack_size++;
                sets0[h0].xormask ^= hash;
                sets0[h0].count--;
                if (sets0[h0].count == 1) {
                    Q0[Q0size].index = h0;
                    Q0[Q0size].hash = sets0[h0].xormask;
                    Q0size++;
                }
                sets2[h2].xormask ^= hash;
                sets2[h2].count--;
                if (sets2[h2].count == 1) {
                    Q2[Q2size].index = h2;
                    Q2[Q2size].hash = sets2[h2].xormask;
                    Q2size++;
                }
            }
            while (Q2size > 0) {
                xor_keyindex_t keyindex = Q2[--Q2size];
                size_t index = keyindex.index;
                if (sets2[index].count == 0)
                    continue;

                //sets2[index].count = 0;
                uint64_t hash = keyindex.hash;

                uint32_t h0 = xor8_get_h0(hash, filter);
                uint32_t h1 = xor8_get_h1(hash, filter);
                keyindex.index += 2 * blockLength;

                stack[stack_size] = keyindex;
                stack_size++;
                sets0[h0].xormask ^= hash;
                sets0[h0].count--;
                if (sets0[h0].count == 1) {
                    Q0[Q0size].index = h0;
                    Q0[Q0size].hash = sets0[h0].xormask;
                    Q0size++;
                }
                sets1[h1].xormask ^= hash;
                sets1[h1].count--;
                if (sets1[h1].count == 1) {
                    Q1[Q1size].index = h1;
                    Q1[Q1size].hash = sets1[h1].xormask;
                    Q1size++;
                }

            }
        }
        if (stack_size == size) {
            // success
            break;
        }

        filter->seed = xor_rng_splitmix64(&rng_counter);
    }
    uint8_t* fingerprints0 = filter->fingerprints;
    uint8_t* fingerprints1 = filter->fingerprints + blockLength;
    uint8_t* fingerprints2 = filter->fingerprints + 2 * blockLength;

    size_t stack_size = size;
    while (stack_size > 0) {
        xor_keyindex_t ki = stack[--stack_size];
        uint64_t val = xor_fingerprint(ki.hash);
        if (ki.index < blockLength) {
            val ^= fingerprints1[xor8_get_h1(ki.hash, filter)] ^ fingerprints2[xor8_get_h2(ki.hash, filter)];
        }
        else if (ki.index < 2 * blockLength) {
            val ^= fingerprints0[xor8_get_h0(ki.hash, filter)] ^ fingerprints2[xor8_get_h2(ki.hash, filter)];
        }
        else {
            val ^= fingerprints0[xor8_get_h0(ki.hash, filter)] ^ fingerprints1[xor8_get_h1(ki.hash, filter)];
        }
        filter->fingerprints[ki.index] = val;
    }

    free(sets);
    free(Q);
    free(stack);
    return true;
}

// Report if the key is in the set, with false positive rate.
static inline bool xor8_contain(uint64_t key, const xor8_t* filter) {
    uint64_t hash = xor_mix_split(key, filter->seed);
    uint8_t f = xor_fingerprint(hash);
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)xor_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)xor_rotl64(hash, 42);
    uint32_t h0 = xor_reduce(r0, filter->blockLength);
    uint32_t h1 = xor_reduce(r1, filter->blockLength) + filter->blockLength;
    uint32_t h2 = xor_reduce(r2, filter->blockLength) + 2 * filter->blockLength;
    return f == (filter->fingerprints[h0] ^ filter->fingerprints[h1] ^
        filter->fingerprints[h2]);
}

bool testxor8(size_t size) {
    printf("testing xor8 ");
    printf("size = %zu \n", size);

    xor8_t filter;

    xor8_allocate(size, &filter);
    // we need some set of values
    uint64_t* big_set = (uint64_t*)malloc(sizeof(uint64_t) * size);
    for (size_t i = 0; i < size; i++) {
        big_set[i] = i; // we use contiguous values
    }
    // we construct the filter
    xor8_populate(big_set, size, &filter); // warm the cache
    for (size_t times = 0; times < 5; times++) {
        clock_t t;
        t = clock();
        xor8_populate(big_set, size, &filter);
        t = clock() - t;
        double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to build an index over %zu values. \n",
            time_taken, size);
    }
    xor8_free(&filter);
    free(big_set);
    return true;
}

bool testxor8() {
    printf("testing xor8\n");

    xor8_t filter;
    size_t size = 10000;
    xor8_allocate(size, &filter);
    // we need some set of values
    uint64_t* big_set = (uint64_t*)malloc(sizeof(uint64_t) * size);
    for (size_t i = 0; i < size; i++) {
        big_set[i] = i; // we use contiguous values
    }
    // we construct the filter
    xor8_populate(big_set, size, &filter);
    for (size_t i = 0; i < size; i++) {
        if (!xor8_contain(big_set[i], &filter)) {
            printf("bug!\n");
            return false;
        }
    }

    size_t random_matches = 0;
    size_t trials = 10000000; //(uint64_t)rand() << 32 + rand()
    for (size_t i = 0; i < trials; i++) {
        uint64_t random_key = ((uint64_t)rand() << 32) + rand();
        if (xor8_contain(random_key, &filter)) {
            if (random_key >= size) {
                random_matches++;
            }
        }
    }
    printf("fpp %3.10f (estimated) \n", random_matches * 1.0 / trials);
    printf("bits per entry %3.1f\n", xor8_size_in_bytes(&filter) * 8.0 / size);
    xor8_free(&filter);
    free(big_set);
    return true;
}

int main() {
    for (size_t s = 10000000; s <= 10000000; s *= 10) {
        // testfuse8(s);
        // testbufferedxor8(s);
        // testxor8(s);
        // testbufferedxor16(s);
        // testxor16(s);
        testxor8();

        printf("\n");
    }
}