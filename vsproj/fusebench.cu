#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define FUSE_ARITY 3
#define FUSE_SEGMENT_COUNT 100
#define FUSE_SLOTS (FUSE_SEGMENT_COUNT + FUSE_ARITY - 1)

#ifndef XOR_MAX_ITERATIONS
#define XOR_MAX_ITERATIONS 100 // probabillity of success should always be > 0.5 so 100 iterations is highly unlikely
#endif 
/**
 * fuse8 is the recommended default, no more than
 * a 0.3% false-positive probability.
 */
typedef struct fuse8_s {
    uint64_t seed;
    uint64_t segmentLength; // = slotCount  / FUSE_SLOTS
    uint8_t
        * fingerprints; // after fuse8_allocate, will point to 3*blockLength values
} fuse8_t;

struct fuse_fuseset_s {
    uint64_t fusemask;
    uint32_t count;
};

typedef struct fuse_fuseset_s fuse_fuseset_t;

struct fuse_keyindex_s {
    uint64_t hash;
    uint32_t index;
};

typedef struct fuse_keyindex_s fuse_keyindex_t;

struct fuse_hashes_s {
    uint64_t h;
    uint32_t h0;
    uint32_t h1;
    uint32_t h2;
};

typedef struct fuse_hashes_s fuse_hashes_t;

struct fuse_h0h1h2_s {
    uint32_t h0;
    uint32_t h1;
    uint32_t h2;
};

typedef struct fuse_h0h1h2_s fuse_h0h1h2_t;

static inline uint64_t fuse_murmur64(uint64_t h) {
    h ^= h >> 33;
    h *= UINT64_C(0xff51afd7ed558ccd);
    h ^= h >> 33;
    h *= UINT64_C(0xc4ceb9fe1a85ec53);
    h ^= h >> 33;
    return h;
}

static inline uint64_t fuse_rotl64(uint64_t n, unsigned int c) {
    return (n << (c & 63)) | (n >> ((-c) & 63));
}

static inline uint32_t fuse_reduce(uint32_t hash, uint32_t n) {
    // http://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
    return (uint32_t)(((uint64_t)hash * n) >> 32);
}

// returns random number, modifies the seed
static inline uint64_t fuse_rng_splitmix64(uint64_t* seed) {
    uint64_t z = (*seed += UINT64_C(0x9E3779B97F4A7C15));
    z = (z ^ (z >> 30)) * UINT64_C(0xBF58476D1CE4E5B9);
    z = (z ^ (z >> 27)) * UINT64_C(0x94D049BB133111EB);
    return z ^ (z >> 31);
}

static inline uint64_t fuse_fingerprint(uint64_t hash) {
    return hash ^ (hash >> 32);
}

// report memory usage
static inline size_t fuse8_size_in_bytes(const fuse8_t* filter) {
    return FUSE_SLOTS * filter->segmentLength * sizeof(uint8_t) + sizeof(fuse8_t);
}

static inline uint64_t fuse_mix_split(uint64_t key, uint64_t seed) {
    return fuse_murmur64(key + seed);
}

static inline fuse_hashes_t fuse8_get_h0_h1_h2(uint64_t k, const fuse8_t* filter) {
    uint64_t hash = fuse_mix_split(k, filter->seed);
    fuse_hashes_t answer;
    answer.h = hash;
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)fuse_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)fuse_rotl64(hash, 42);
    uint32_t r3 = (0xBF58476D1CE4E5B9 * hash) >> 32;
    uint32_t seg = fuse_reduce(r0, FUSE_SEGMENT_COUNT);
    answer.h0 = (seg + 0) * filter->segmentLength + fuse_reduce(r1, filter->segmentLength);
    answer.h1 = (seg + 1) * filter->segmentLength + fuse_reduce(r2, filter->segmentLength);
    answer.h2 = (seg + 2) * filter->segmentLength + fuse_reduce(r3, filter->segmentLength);
    return answer;
}

static inline fuse_h0h1h2_t fuse8_get_just_h0_h1_h2(uint64_t hash,
    const fuse8_t* filter) {
    fuse_h0h1h2_t answer;
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)fuse_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)fuse_rotl64(hash, 42);
    uint32_t r3 = (0xBF58476D1CE4E5B9 * hash) >> 32;
    uint32_t seg = fuse_reduce(r0, FUSE_SEGMENT_COUNT);
    answer.h0 = (seg + 0) * filter->segmentLength + fuse_reduce(r1, filter->segmentLength);
    answer.h1 = (seg + 1) * filter->segmentLength + fuse_reduce(r2, filter->segmentLength);
    answer.h2 = (seg + 2) * filter->segmentLength + fuse_reduce(r3, filter->segmentLength);
    return answer;
}

// allocate enough capacity for a set containing up to 'size' elements
// caller is responsible to call fuse8_free(filter)
static inline bool fuse8_allocate(uint32_t size, fuse8_t* filter) {
    size_t capacity = 1.0 / 0.879 * size;
    capacity = capacity / FUSE_SLOTS * FUSE_SLOTS;
    filter->fingerprints = (uint8_t*)malloc(capacity * sizeof(uint8_t));
    if (filter->fingerprints != NULL) {
        filter->segmentLength = capacity / FUSE_SLOTS;
        return true;
    }
    else {
        return false;
    }
}

bool fuse8_populate(const uint64_t* keys, uint32_t size, fuse8_t* filter) {
    uint64_t rng_counter = 1;
    filter->seed = fuse_rng_splitmix64(&rng_counter);
    size_t arrayLength = filter->segmentLength * FUSE_SLOTS; // size of the backing array
    //size_t segmentLength = filter->segmentLength;
    fuse_fuseset_t* sets =
        (fuse_fuseset_t*)malloc(arrayLength * sizeof(fuse_fuseset_t));

    fuse_keyindex_t* Q =
        (fuse_keyindex_t*)malloc(arrayLength * sizeof(fuse_keyindex_t));

    fuse_keyindex_t* stack =
        (fuse_keyindex_t*)malloc(size * sizeof(fuse_keyindex_t));

    if ((sets == NULL) || (Q == NULL) || (stack == NULL)) {
        free(sets);
        free(Q);
        free(stack);
        return false;
    }

    for (int loop = 0; true; ++loop) {
        if (loop + 1 > XOR_MAX_ITERATIONS) {
            fprintf(stderr, "Too many iterations. Are all your keys unique?");
            free(sets);
            free(Q);
            free(stack);
            return false;
        }


        memset(sets, 0, sizeof(fuse_fuseset_t) * arrayLength);
        for (size_t i = 0; i < size; i++) {
            uint64_t key = keys[i];
            fuse_hashes_t hs = fuse8_get_h0_h1_h2(key, filter);
            sets[hs.h0].fusemask ^= hs.h;
            sets[hs.h0].count++;
            sets[hs.h1].fusemask ^= hs.h;
            sets[hs.h1].count++;
            sets[hs.h2].fusemask ^= hs.h;
            sets[hs.h2].count++;
        }
        // todo: the flush should be sync with the detection that follows
        // scan for values with a count of one
        size_t Qsize = 0;
        for (size_t i = 0; i < arrayLength; i++) {
            if (sets[i].count == 1) {
                Q[Qsize].index = i;
                Q[Qsize].hash = sets[i].fusemask;
                Qsize++;
            }
        }

        size_t stack_size = 0;
        while (Qsize > 0) {
            fuse_keyindex_t keyindex = Q[--Qsize];
            size_t index = keyindex.index;
            if (sets[index].count == 0)
                continue;  // not actually possible after the initial scan.
              // sets0[index].count = 0;
            uint64_t hash = keyindex.hash;
            fuse_h0h1h2_t hs = fuse8_get_just_h0_h1_h2(hash, filter);

            stack[stack_size] = keyindex;
            stack_size++;

            //if (hs.h0 != index) {
            sets[hs.h0].fusemask ^= hash;
            sets[hs.h0].count--;
            if (sets[hs.h0].count == 1) {
                Q[Qsize].index = hs.h0;
                Q[Qsize].hash = sets[hs.h0].fusemask;
                Qsize++;
            }
            //}

            //if (hs.h1 != index) {
            sets[hs.h1].fusemask ^= hash;
            sets[hs.h1].count--;
            if (sets[hs.h1].count == 1) {
                Q[Qsize].index = hs.h1;
                Q[Qsize].hash = sets[hs.h1].fusemask;
                Qsize++;
            }
            //}

            //if (hs.h2 != index) {
            sets[hs.h2].fusemask ^= hash;
            sets[hs.h2].count--;
            if (sets[hs.h2].count == 1) {
                Q[Qsize].index = hs.h2;
                Q[Qsize].hash = sets[hs.h2].fusemask;
                Qsize++;
            }
            //}
        }

        if (stack_size == size) {
            // success
            break;
        }

        filter->seed = fuse_rng_splitmix64(&rng_counter);
    }

    size_t stack_size = size;
    while (stack_size > 0) {
        fuse_keyindex_t ki = stack[--stack_size];
        fuse_h0h1h2_t hs = fuse8_get_just_h0_h1_h2(ki.hash, filter);
        uint8_t hsh = fuse_fingerprint(ki.hash);
        if (ki.index == hs.h0) {
            hsh ^= filter->fingerprints[hs.h1] ^ filter->fingerprints[hs.h2];
        }
        else if (ki.index == hs.h1) {
            hsh ^= filter->fingerprints[hs.h0] ^ filter->fingerprints[hs.h2];
        }
        else {
            hsh ^= filter->fingerprints[hs.h0] ^ filter->fingerprints[hs.h1];
        }
        filter->fingerprints[ki.index] = hsh;
    }

    free(sets);
    free(Q);
    free(stack);
    return true;
}

static inline bool fuse8_contain(uint64_t key, const fuse8_t* filter) {
    uint64_t hash = fuse_mix_split(key, filter->seed);
    uint8_t f = fuse_fingerprint(hash);
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)fuse_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)fuse_rotl64(hash, 42);
    uint32_t r3 = (0xBF58476D1CE4E5B9 * hash) >> 32;
    uint32_t seg = fuse_reduce(r0, FUSE_SEGMENT_COUNT);
    uint32_t h0 = (seg + 0) * filter->segmentLength + fuse_reduce(r1, filter->segmentLength);
    uint32_t h1 = (seg + 1) * filter->segmentLength + fuse_reduce(r2, filter->segmentLength);
    uint32_t h2 = (seg + 2) * filter->segmentLength + fuse_reduce(r3, filter->segmentLength);
    return f == (filter->fingerprints[h0] ^ filter->fingerprints[h1] ^
        filter->fingerprints[h2]);
}

// release memory
static inline void fuse8_free(fuse8_t* filter) {
    free(filter->fingerprints);
    filter->fingerprints = NULL;
    filter->segmentLength = 0;
}

bool testfuse8() {
    printf("testing fuse8\n");

    fuse8_t filter;
    // cudaMallocManaged(&filter, sizeof(fuse8_t));

    size_t size = 1000000;
    fuse8_allocate(size, &filter);
    // we need some set of values
    uint64_t* big_set = (uint64_t*)malloc(sizeof(uint64_t) * size);
    for (size_t i = 0; i < size; i++) {
        big_set[i] = i; // we use contiguous values
    }
    // we construct the filter
    fuse8_populate(big_set, size, &filter);
    for (size_t i = 0; i < size; i++) {
        if (!fuse8_contain(big_set[i], &filter)) {
            printf("bug!\n");
            return false;
        }
    }

    size_t random_matches = 0;
    size_t trials = 10000000; //(uint64_t)rand() << 32 + rand()
    for (size_t i = 0; i < trials; i++) {
        uint64_t random_key = ((uint64_t)rand() << 32) + rand();
        if (fuse8_contain(random_key, &filter)) {
            if (random_key >= size) {
                random_matches++;
            }
        }
    }
    printf("fpp %3.10f (estimated) \n", random_matches * 1.0 / trials);
    printf("bits per entry %3.1f\n", fuse8_size_in_bytes(&filter) * 8.0 / size);
    fuse8_free(&filter);
    free(big_set);
    return true;
}

bool testfuse8(size_t size) {
    printf("testing fuse8 ");
    printf("size = %zu \n", size);

    fuse8_t filter;

    fuse8_allocate(size, &filter);
    // we need some set of values
    uint64_t* big_set = (uint64_t*)malloc(sizeof(uint64_t) * size);
    for (size_t i = 0; i < size; i++) {
        big_set[i] = i; // we use contiguous values
    }
    // we construct the filter
    fuse8_populate(big_set, size, &filter); // warm the cache
    for (size_t times = 0; times < 5; times++) {
        clock_t t;
        t = clock();
        fuse8_populate(big_set, size, &filter);
        t = clock() - t;
        double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to build an index over %zu values. \n",
            time_taken, size);
    }
    fuse8_free(&filter);
    free(big_set);
    return true;
}

int main() {
    for (size_t s = 10000000; s <= 10000000; s *= 10) {
        // testfuse8(s);
        // testbufferedxor8(s);
        // testxor8(s);
        // testbufferedxor16(s);
        // testxor16(s);
        // testfuse8();

        printf("\n");
    }
}