#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <>

#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#define FUSE_ARITY 3
#define FUSE_SEGMENT_COUNT 100
#define FUSE_SLOTS (FUSE_SEGMENT_COUNT + FUSE_ARITY - 1)

#ifndef XOR_MAX_ITERATIONS
#define XOR_MAX_ITERATIONS 100 // probabillity of success should always be > 0.5 so 100 iterations is highly unlikely
#endif 
/**
 * fuse8 is the recommended default, no more than
 * a 0.3% false-positive probability.
 */
typedef struct fuse8_s {
    uint64_t seed;
    uint64_t segmentLength; // = slotCount  / FUSE_SLOTS
    uint8_t
        * fingerprints; // after fuse8_allocate, will point to 3*blockLength values
} fuse8_t;

struct fuse_fuseset_s {
    // uint64_t fusemask;
    uint32_t fusemask1;
    uint32_t fusemask2;
    uint32_t count;
    uint32_t layer;
    uint32_t origIdx; // store original index before sorting
};

typedef struct fuse_fuseset_s fuse_fuseset_t;

struct fuse_keyindex_s {
    uint64_t hash;
    uint32_t index;
};

typedef struct fuse_keyindex_s fuse_keyindex_t;

struct fuse_hashes_s {
    uint64_t h;
    uint32_t h0;
    uint32_t h1;
    uint32_t h2;
};

typedef struct fuse_hashes_s fuse_hashes_t;

struct fuse_h0h1h2_s {
    uint32_t h0;
    uint32_t h1;
    uint32_t h2;
};

typedef struct fuse_h0h1h2_s fuse_h0h1h2_t;

struct is_recovered {
    __host__ __device__
    bool operator()(fuse_fuseset_t &x) {
        return x.layer > 0;
    }
};

static inline uint64_t fuse_murmur64(uint64_t h) {
    h ^= h >> 33;
    h *= UINT64_C(0xff51afd7ed558ccd);
    h ^= h >> 33;
    h *= UINT64_C(0xc4ceb9fe1a85ec53);
    h ^= h >> 33;
    return h;
}

static inline uint64_t fuse_rotl64(uint64_t n, unsigned int c) {
    return (n << (c & 63)) | (n >> ((-c) & 63));
}

static inline uint32_t fuse_reduce(uint32_t hash, uint32_t n) {
    // http://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
    return (uint32_t)(((uint64_t)hash * n) >> 32);
}

// returns random number, modifies the seed
static inline uint64_t fuse_rng_splitmix64(uint64_t* seed) {
    uint64_t z = (*seed += UINT64_C(0x9E3779B97F4A7C15));
    z = (z ^ (z >> 30)) * UINT64_C(0xBF58476D1CE4E5B9);
    z = (z ^ (z >> 27)) * UINT64_C(0x94D049BB133111EB);
    return z ^ (z >> 31);
}

static inline uint64_t fuse_fingerprint(uint64_t hash) {
    return hash ^ (hash >> 32);
}

// report memory usage
static inline size_t fuse8_size_in_bytes(const fuse8_t* filter) {
    return FUSE_SLOTS * filter->segmentLength * sizeof(uint8_t) + sizeof(fuse8_t);
}

static inline uint64_t fuse_mix_split(uint64_t key, uint64_t seed) {
    return fuse_murmur64(key + seed);
}

__device__
static inline uint64_t d_fuse_murmur64(uint64_t h) {
    h ^= h >> 33;
    h *= UINT64_C(0xff51afd7ed558ccd);
    h ^= h >> 33;
    h *= UINT64_C(0xc4ceb9fe1a85ec53);
    h ^= h >> 33;
    return h;
}

__device__
static inline uint64_t d_fuse_fingerprint(uint64_t hash) {
    return hash ^ (hash >> 32);
}

__device__
static inline uint64_t d_fuse_rotl64(uint64_t n, unsigned int c) {
    return (n << (c & 63)) | (n >> ((-c) & 63));
}

__device__
static inline uint32_t d_fuse_reduce(uint32_t hash, uint32_t n) {
    // http://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
    return (uint32_t)(((uint64_t)hash * n) >> 32);
}

__device__
static inline uint64_t d_fuse_mix_split(uint64_t key, uint64_t seed) {
    return d_fuse_murmur64(key + seed);
}

__device__
static inline fuse_hashes_t d_fuse8_get_h0_h1_h2(uint64_t k, const fuse8_t* filter) {
    uint64_t hash = d_fuse_mix_split(k, filter->seed);
    fuse_hashes_t answer;
    answer.h = hash;
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)d_fuse_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)d_fuse_rotl64(hash, 42);
    uint32_t r3 = (0xBF58476D1CE4E5B9 * hash) >> 32;
    uint32_t seg = d_fuse_reduce(r0, FUSE_SEGMENT_COUNT);
    answer.h0 = (seg + 0) * filter->segmentLength + d_fuse_reduce(r1, filter->segmentLength);
    answer.h1 = (seg + 1) * filter->segmentLength + d_fuse_reduce(r2, filter->segmentLength);
    answer.h2 = (seg + 2) * filter->segmentLength + d_fuse_reduce(r3, filter->segmentLength);
    return answer;
}

__device__
static inline fuse_h0h1h2_t d_fuse8_get_just_h0_h1_h2(uint64_t hash,
    const fuse8_t* filter) {
    fuse_h0h1h2_t answer;
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)d_fuse_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)d_fuse_rotl64(hash, 42);
    uint32_t r3 = (0xBF58476D1CE4E5B9 * hash) >> 32;
    uint32_t seg = d_fuse_reduce(r0, FUSE_SEGMENT_COUNT);
    answer.h0 = (seg + 0) * filter->segmentLength + d_fuse_reduce(r1, filter->segmentLength);
    answer.h1 = (seg + 1) * filter->segmentLength + d_fuse_reduce(r2, filter->segmentLength);
    answer.h2 = (seg + 2) * filter->segmentLength + d_fuse_reduce(r3, filter->segmentLength);
    return answer;
}

__global__
void insert_keys(const uint64_t* keys, uint32_t size, fuse_fuseset_t* sets, fuse8_t* filter) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride) {
        uint64_t key = keys[i];
        fuse_hashes_t hs = d_fuse8_get_h0_h1_h2(key, filter);
        uint32_t hsh1 = (uint32_t)(hs.h >> 32);
        uint32_t hsh2 = (uint32_t)hs.h;
        atomicXor(&sets[hs.h0].fusemask1, hsh1);
        atomicXor(&sets[hs.h0].fusemask2, hsh2);
        atomicAdd(&sets[hs.h0].count, 1);
        atomicXor(&sets[hs.h1].fusemask1, hsh1);
        atomicXor(&sets[hs.h1].fusemask2, hsh2);
        atomicAdd(&sets[hs.h1].count, 1);
        atomicXor(&sets[hs.h2].fusemask1, hsh1);
        atomicXor(&sets[hs.h2].fusemask2, hsh2);
        atomicAdd(&sets[hs.h2].count, 1);
    }
}

__global__
void peel_set0(fuse_fuseset_t* sets, fuse_fuseset_t* sets0, fuse_fuseset_t* sets1, 
    fuse_fuseset_t* sets2, fuse8_t* filter, size_t* layer, bool* pureCell) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < filter->segmentLength * (FUSE_SLOTS / 3); i += stride) {
        if (sets0[i].count == 1) {
            uint32_t sets_idx = ((i / filter->segmentLength) * 3) * filter->segmentLength + i % filter->segmentLength;
            sets0[i].count--;
            sets[sets_idx].count--;
            uint64_t hash = ((uint64_t)sets0[i].fusemask1) << 32 | sets0[i].fusemask2;
            sets0[i].layer = *layer;
            sets[sets_idx].layer = *layer;
            // store the original index, in case the array is sorted
            sets[sets_idx].origIdx = sets_idx;
            *pureCell = true; // race condition but should be safe
            fuse_h0h1h2_t hs = d_fuse8_get_just_h0_h1_h2(hash, filter);

            if (hs.h0 != sets_idx) {
                atomicXor(&sets[hs.h0].fusemask1, sets0[i].fusemask1);
                atomicXor(&sets[hs.h0].fusemask2, sets0[i].fusemask2);
                atomicSub(&sets[hs.h0].count, 1);

                uint32_t h0_seg = hs.h0 / filter->segmentLength;
                uint32_t h0_idx = (h0_seg / 3) * filter->segmentLength + hs.h0 % filter->segmentLength;
                fuse_fuseset_t* h0_set;
                if (h0_seg % FUSE_ARITY == 0) {
                    h0_set = sets0;
                }
                else if (h0_seg % FUSE_ARITY == 1) {
                    h0_set = sets1;
                }
                else {
                    h0_set = sets2;
                }
                atomicXor(&h0_set[h0_idx].fusemask1, sets0[i].fusemask1);
                atomicXor(&h0_set[h0_idx].fusemask2, sets0[i].fusemask2);
                atomicSub(&h0_set[h0_idx].count, 1);
            }

            if (hs.h1 != sets_idx) {
                atomicXor(&sets[hs.h1].fusemask1, sets0[i].fusemask1);
                atomicXor(&sets[hs.h1].fusemask2, sets0[i].fusemask2);
                atomicSub(&sets[hs.h1].count, 1);

                uint32_t h1_seg = hs.h1 / filter->segmentLength;
                uint32_t h1_idx = (h1_seg / 3) * filter->segmentLength + hs.h1 % filter->segmentLength;
                fuse_fuseset_t* h1_set;
                if (h1_seg % FUSE_ARITY == 0) {
                    h1_set = sets0;
                }
                else if (h1_seg % FUSE_ARITY == 1) {
                    h1_set = sets1;
                }
                else {
                    h1_set = sets2;
                }
                atomicXor(&h1_set[h1_idx].fusemask1, sets0[i].fusemask1);
                atomicXor(&h1_set[h1_idx].fusemask2, sets0[i].fusemask2);
                atomicSub(&h1_set[h1_idx].count, 1);
            }

            if (hs.h2 != sets_idx) {
                atomicXor(&sets[hs.h2].fusemask1, sets0[i].fusemask1);
                atomicXor(&sets[hs.h2].fusemask2, sets0[i].fusemask2);
                atomicSub(&sets[hs.h2].count, 1);

                uint32_t h2_seg = hs.h2 / filter->segmentLength;
                uint32_t h2_idx = (h2_seg / 3) * filter->segmentLength + hs.h2 % filter->segmentLength;
                fuse_fuseset_t* h2_set;
                if (h2_seg % FUSE_ARITY == 0) {
                    h2_set = sets0;
                }
                else if (h2_seg % FUSE_ARITY == 1) {
                    h2_set = sets1;
                }
                else {
                    h2_set = sets2;
                }
                atomicXor(&h2_set[h2_idx].fusemask1, sets0[i].fusemask1);
                atomicXor(&h2_set[h2_idx].fusemask2, sets0[i].fusemask2);
                atomicSub(&h2_set[h2_idx].count, 1);
            }

        }
    }
}

__global__
void peel_set1(fuse_fuseset_t* sets, fuse_fuseset_t* sets0, fuse_fuseset_t* sets1,
    fuse_fuseset_t* sets2, fuse8_t* filter, size_t* layer, bool* pureCell) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < filter->segmentLength * (FUSE_SLOTS / 3); i += stride) {
        if (sets1[i].count == 1) {
            uint32_t sets_idx = ((i / filter->segmentLength) * 3 + 1) * filter->segmentLength + i % filter->segmentLength;
            sets1[i].count--;
            sets[sets_idx].count--;
            uint64_t hash = ((uint64_t)sets1[i].fusemask1) << 32 | sets1[i].fusemask2;
            sets1[i].layer = *layer;
            sets[sets_idx].layer = *layer;
            // store the original index, in case the array is sorted
            sets[sets_idx].origIdx = sets_idx;
            *pureCell = true; // race condition but should be safe
            fuse_h0h1h2_t hs = d_fuse8_get_just_h0_h1_h2(hash, filter);

            if (hs.h0 != sets_idx) {
                atomicXor(&sets[hs.h0].fusemask1, sets1[i].fusemask1);
                atomicXor(&sets[hs.h0].fusemask2, sets1[i].fusemask2);
                atomicSub(&sets[hs.h0].count, 1);

                uint32_t h0_seg = hs.h0 / filter->segmentLength;
                uint32_t h0_idx = (h0_seg / 3) * filter->segmentLength + hs.h0 % filter->segmentLength;
                fuse_fuseset_t* h0_set;
                if (h0_seg % FUSE_ARITY == 0) {
                    h0_set = sets0;
                }
                else if (h0_seg % FUSE_ARITY == 1) {
                    h0_set = sets1;
                }
                else {
                    h0_set = sets2;
                }
                atomicXor(&h0_set[h0_idx].fusemask1, sets1[i].fusemask1);
                atomicXor(&h0_set[h0_idx].fusemask2, sets1[i].fusemask2);
                atomicSub(&h0_set[h0_idx].count, 1);
            }

            if (hs.h1 != sets_idx) {
                atomicXor(&sets[hs.h1].fusemask1, sets1[i].fusemask1);
                atomicXor(&sets[hs.h1].fusemask2, sets1[i].fusemask2);
                atomicSub(&sets[hs.h1].count, 1);

                uint32_t h1_seg = hs.h1 / filter->segmentLength;
                uint32_t h1_idx = (h1_seg / 3) * filter->segmentLength + hs.h1 % filter->segmentLength;
                fuse_fuseset_t* h1_set;
                if (h1_seg % FUSE_ARITY == 0) {
                    h1_set = sets0;
                }
                else if (h1_seg % FUSE_ARITY == 1) {
                    h1_set = sets1;
                }
                else {
                    h1_set = sets2;
                }
                atomicXor(&h1_set[h1_idx].fusemask1, sets1[i].fusemask1);
                atomicXor(&h1_set[h1_idx].fusemask2, sets1[i].fusemask2);
                atomicSub(&h1_set[h1_idx].count, 1);
            }

            if (hs.h2 != sets_idx) {
                atomicXor(&sets[hs.h2].fusemask1, sets1[i].fusemask1);
                atomicXor(&sets[hs.h2].fusemask2, sets1[i].fusemask2);
                atomicSub(&sets[hs.h2].count, 1);

                uint32_t h2_seg = hs.h2 / filter->segmentLength;
                uint32_t h2_idx = (h2_seg / 3) * filter->segmentLength + hs.h2 % filter->segmentLength;
                fuse_fuseset_t* h2_set;
                if (h2_seg % FUSE_ARITY == 0) {
                    h2_set = sets0;
                }
                else if (h2_seg % FUSE_ARITY == 1) {
                    h2_set = sets1;
                }
                else {
                    h2_set = sets2;
                }
                atomicXor(&h2_set[h2_idx].fusemask1, sets1[i].fusemask1);
                atomicXor(&h2_set[h2_idx].fusemask2, sets1[i].fusemask2);
                atomicSub(&h2_set[h2_idx].count, 1);
            }

        }
    }
}

__global__
void peel_set2(fuse_fuseset_t* sets, fuse_fuseset_t* sets0, fuse_fuseset_t* sets1,
    fuse_fuseset_t* sets2, fuse8_t* filter, size_t* layer, bool* pureCell) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < filter->segmentLength * (FUSE_SLOTS / 3); i += stride) {
        if (sets2[i].count == 1) {
            uint32_t sets_idx = ((i / filter->segmentLength) * 3 + 2) * filter->segmentLength + i % filter->segmentLength;
            sets2[i].count--;
            sets[sets_idx].count--;
            uint64_t hash = ((uint64_t)sets2[i].fusemask1) << 32 | sets2[i].fusemask2;
            sets2[i].layer = *layer;
            sets[sets_idx].layer = *layer;
            // store the original index, in case the array is sorted
            sets[sets_idx].origIdx = sets_idx;
            *pureCell = true; // race condition but should be safe
            fuse_h0h1h2_t hs = d_fuse8_get_just_h0_h1_h2(hash, filter);

            if (hs.h0 != sets_idx) {
                atomicXor(&sets[hs.h0].fusemask1, sets2[i].fusemask1);
                atomicXor(&sets[hs.h0].fusemask2, sets2[i].fusemask2);
                atomicSub(&sets[hs.h0].count, 1);

                uint32_t h0_seg = hs.h0 / filter->segmentLength;
                uint32_t h0_idx = (h0_seg / 3) * filter->segmentLength + hs.h0 % filter->segmentLength;
                fuse_fuseset_t* h0_set;
                if (h0_seg % FUSE_ARITY == 0) {
                    h0_set = sets0;
                }
                else if (h0_seg % FUSE_ARITY == 1) {
                    h0_set = sets1;
                }
                else {
                    h0_set = sets2;
                }
                atomicXor(&h0_set[h0_idx].fusemask1, sets2[i].fusemask1);
                atomicXor(&h0_set[h0_idx].fusemask2, sets2[i].fusemask2);
                atomicSub(&h0_set[h0_idx].count, 1);
            }

            if (hs.h1 != sets_idx) {
                atomicXor(&sets[hs.h1].fusemask1, sets2[i].fusemask1);
                atomicXor(&sets[hs.h1].fusemask2, sets2[i].fusemask2);
                atomicSub(&sets[hs.h1].count, 1);

                uint32_t h1_seg = hs.h1 / filter->segmentLength;
                uint32_t h1_idx = (h1_seg / 3) * filter->segmentLength + hs.h1 % filter->segmentLength;
                fuse_fuseset_t* h1_set;
                if (h1_seg % FUSE_ARITY == 0) {
                    h1_set = sets0;
                }
                else if (h1_seg % FUSE_ARITY == 1) {
                    h1_set = sets1;
                }
                else {
                    h1_set = sets2;
                }
                atomicXor(&h1_set[h1_idx].fusemask1, sets2[i].fusemask1);
                atomicXor(&h1_set[h1_idx].fusemask2, sets2[i].fusemask2);
                atomicSub(&h1_set[h1_idx].count, 1);
            }

            if (hs.h2 != sets_idx) {
                atomicXor(&sets[hs.h2].fusemask1, sets2[i].fusemask1);
                atomicXor(&sets[hs.h2].fusemask2, sets2[i].fusemask2);
                atomicSub(&sets[hs.h2].count, 1);

                uint32_t h2_seg = hs.h2 / filter->segmentLength;
                uint32_t h2_idx = (h2_seg / 3) * filter->segmentLength + hs.h2 % filter->segmentLength;
                fuse_fuseset_t* h2_set;
                if (h2_seg % FUSE_ARITY == 0) {
                    h2_set = sets0;
                }
                else if (h2_seg % FUSE_ARITY == 1) {
                    h2_set = sets1;
                }
                else {
                    h2_set = sets2;
                }
                atomicXor(&h2_set[h2_idx].fusemask1, sets2[i].fusemask1);
                atomicXor(&h2_set[h2_idx].fusemask2, sets2[i].fusemask2);
                atomicSub(&h2_set[h2_idx].count, 1);
            }

        }
    }
}

static inline int compare_layers(const void* p, const void* q) {
    uint32_t x = (*(const fuse_fuseset_t*)p).layer;
    uint32_t y = (*(const fuse_fuseset_t*)q).layer;

    // Avoid return x - y, which can cause undefined behaviour
    //   because of signed integer overflow.
    if (x < y)
        return 1;
    else if (x > y)
        return -1;
    return 0;
}

static inline uint32_t* sortcount_layers(fuse_fuseset_t* sets, size_t n, size_t max_layer) {
    uint32_t* keys;
    hipError_t err_keys = hipMallocManaged(&keys, n * sizeof(uint32_t));

    if (err_keys != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified vector (error code %s)!\n", hipGetErrorString(err_keys));
        return NULL;
    }

    for (size_t i = 0; i < n; i++)
        keys[i] = sets[i].layer;

    thrust::sort_by_key(thrust::device, keys, keys + n, sets, thrust::greater<uint32_t>());
    hipDeviceSynchronize();
    thrust::pair<uint32_t*, uint32_t*> new_end;
        
    uint32_t* hist_keys;                         
    uint32_t* hist_freqs;                         
    hipError_t err_hist_keys = hipMallocManaged(&hist_keys, max_layer * sizeof(uint32_t));
    hipError_t err_hist_freqs = hipMallocManaged(&hist_freqs, max_layer * sizeof(uint32_t));

    if (err_hist_keys != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified vector (error code %s)!\n", hipGetErrorString(err_hist_keys));
        return NULL;
    }

    if (err_hist_freqs != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified vector (error code %s)!\n", hipGetErrorString(err_hist_freqs));
        return NULL;
    }

    new_end = thrust::reduce_by_key(thrust::device, keys, keys + n, thrust::make_constant_iterator(1), hist_keys, hist_freqs);
    hipDeviceSynchronize();

    hipFree(keys);
    hipFree(hist_keys);

    return hist_freqs;
}

// TODO Remove temporary method to generate a histogram of layer keys
static inline uint32_t * count_by_key(fuse_fuseset_t* sets, size_t n, size_t max_value) {
    uint32_t* counts;
    // To prevent array OOB, we initialize counting array with size = max_key_value + 1
    hipError_t err_counts = hipMallocManaged(&counts, (max_value+1) * sizeof(uint32_t));
    
    if (err_counts != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified vector (error code %s)!\n", hipGetErrorString(err_counts));
        return NULL;
    }

    // Ensure counting array is zero-ed
    memset(counts, 0, (max_value+1) * sizeof(uint32_t));

    for (size_t i = 0; i < n; i++) {
        counts[sets[i].layer]++;
    }

    return counts;
}

__global__
void assign(fuse8_t* filter, fuse_fuseset_t* sets, size_t layer, size_t arrayLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < arrayLength; i += stride) {
        if (sets[i].layer == layer) {
            uint64_t key_hash = ((uint64_t)sets[i].fusemask1) << 32 | sets[i].fusemask2; // 32-bit limitation on nVidia GPU
            fuse_h0h1h2_t hs = d_fuse8_get_just_h0_h1_h2(key_hash, filter);
            uint8_t hsh = d_fuse_fingerprint(key_hash);

            if (i == hs.h0) {
                hsh ^= filter->fingerprints[hs.h1] ^ filter->fingerprints[hs.h2];
            }
            else if (i == hs.h1) {
                hsh ^= filter->fingerprints[hs.h0] ^ filter->fingerprints[hs.h2];
            }
            else {
                hsh ^= filter->fingerprints[hs.h0] ^ filter->fingerprints[hs.h1];
            }
            filter->fingerprints[i] = hsh;
        }
    }
}

__global__
void assign_compacted(fuse8_t* filter, fuse_fuseset_t* sets, size_t layer, size_t compactLen) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < compactLen; i += stride) {
        uint64_t key_hash = ((uint64_t)sets[i].fusemask1) << 32 | sets[i].fusemask2; // 32-bit limitation on nVidia GPU
        fuse_h0h1h2_t hs = d_fuse8_get_just_h0_h1_h2(key_hash, filter);
        uint8_t hsh = d_fuse_fingerprint(key_hash);

        // TODO revert debugging code
        /* uint32_t h0 = hs.h0;
        uint32_t h1 = hs.h1;
        uint32_t h2 = hs.h2;
        uint32_t orig = sets[i].origIdx;*/ 

        if (sets[i].origIdx == hs.h0) {
            hsh ^= filter->fingerprints[hs.h1] ^ filter->fingerprints[hs.h2];
        }
        else if (sets[i].origIdx == hs.h1) {
            hsh ^= filter->fingerprints[hs.h0] ^ filter->fingerprints[hs.h2];
        }
        else {
            hsh ^= filter->fingerprints[hs.h0] ^ filter->fingerprints[hs.h1];
        }
        filter->fingerprints[sets[i].origIdx] = hsh;
    }
}



// allocate enough capacity for a set containing up to 'size' elements
// caller is responsible to call fuse8_free(filter)
static inline bool fuse8_allocate(uint32_t size, fuse8_t* filter) {
    size_t capacity = 1.0 / 0.879 * size;
    capacity = capacity / FUSE_SLOTS * FUSE_SLOTS;
    hipMallocManaged(&filter->fingerprints, capacity * sizeof(uint8_t));
    if (filter->fingerprints != NULL) {
        filter->segmentLength = capacity / FUSE_SLOTS;
        return true;
    }
    else {
        return false;
    }
}

// TODO Remove temporary method to count number of pure cells per parallel peeling subround
void count_pure_cells(fuse8_t* filter, const fuse_fuseset_t* arr, size_t setId) {
    uint32_t cnt = 0;
    for (uint64_t i = 0; i < filter->segmentLength * (FUSE_SLOTS / FUSE_ARITY); i++) {
        if (arr[i].count == 1) {
            cnt += 1;
        }
    }
    printf("Found %lu pure cells in set%zu\n", cnt, setId);
}

bool fuse8_populate(const uint64_t* keys, uint32_t size, fuse8_t* filter) {
    clock_t setup_t = clock();
    uint64_t rng_counter = 1;
    filter->seed = fuse_rng_splitmix64(&rng_counter);
    size_t arrayLength = filter->segmentLength * FUSE_SLOTS; // size of the backing array
    fuse_fuseset_t* sets;
    hipError_t errSets = hipMallocManaged(&sets, arrayLength * sizeof(fuse_fuseset_t));

    if (errSets != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(errSets));
        return false;
    }

    size_t* layer;
    hipMallocManaged(&layer, sizeof(size_t));
    *layer = 0;

    // Default kernel config
    int blockSize = 128;
    int numBlocks = (filter->segmentLength + blockSize - 1) / blockSize;

    for (int loop = 0; true; ++loop) {
        if (loop + 1 > XOR_MAX_ITERATIONS) {
            fprintf(stderr, "Too many iterations. Are all your keys unique?");
            hipFree(sets);
            return false;
        }

        memset(sets, 0, sizeof(fuse_fuseset_t) * arrayLength);
        setup_t = clock() - setup_t;
        double time_taken = ((double)setup_t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to setup backing array.\n", time_taken);

        clock_t insert_t = clock();
        insert_keys << <numBlocks, blockSize >> > (keys, size, sets, filter);
        hipDeviceSynchronize();
        insert_t = clock() - insert_t;
        time_taken = ((double)insert_t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to insert %u values. \n", time_taken, size);

        fuse_fuseset_t* sets0, * sets1, * sets2;

        clock_t copy_t = clock();
        // copy out sets into 3 smaller arrays
        hipError_t errSets0 = hipMallocManaged(&sets0, filter->segmentLength * (FUSE_SLOTS / FUSE_ARITY) * sizeof(fuse_fuseset_t));
        hipError_t errSets1 = hipMallocManaged(&sets1, filter->segmentLength * (FUSE_SLOTS / FUSE_ARITY) * sizeof(fuse_fuseset_t));
        hipError_t errSets2 = hipMallocManaged(&sets2, filter->segmentLength * (FUSE_SLOTS / FUSE_ARITY) * sizeof(fuse_fuseset_t));

        for (uint32_t seg = 0; seg < (FUSE_SLOTS / FUSE_ARITY); seg++) {
            uint32_t dst0_offset = seg * filter->segmentLength;
            uint32_t src0_offset = seg * FUSE_ARITY * filter->segmentLength;
            memcpy(&sets0[dst0_offset], &sets[src0_offset], filter->segmentLength * sizeof(fuse_fuseset_t));
            uint32_t dst1_offset = seg * filter->segmentLength;
            uint32_t src1_offset = (seg * FUSE_ARITY + 1) * filter->segmentLength;
            memcpy(&sets1[dst1_offset], &sets[src1_offset], filter->segmentLength * sizeof(fuse_fuseset_t));
            uint32_t dst2_offset = seg * filter->segmentLength;
            uint32_t src2_offset = (seg * FUSE_ARITY + 2) * filter->segmentLength;
            memcpy(&sets2[dst2_offset], &sets[src2_offset], filter->segmentLength * sizeof(fuse_fuseset_t));
        }

        copy_t = clock() - copy_t;
        time_taken = ((double)copy_t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to copy subarrays. \n", time_taken);

        bool* pureCell;
        hipMallocManaged(&pureCell, sizeof(bool));
        *pureCell = false;
        *layer = 1;
        size_t old_layer = 0;
        size_t subrounds = 0;

        clock_t peel_t = clock();
        while (old_layer != *layer) { // Did we peel anything over three subrounds
            old_layer = *layer;

            // countPureCells(filter, sets0, 0);
            peel_set0 << <numBlocks, blockSize >> > (sets, sets0, sets1, sets2, filter, layer, pureCell);
            hipDeviceSynchronize();
            subrounds++;

            if (*pureCell == true) {
                *layer = *layer + 1;
                *pureCell = false;
            }

            // countPureCells(filter, sets1, 1);
            peel_set1 << <numBlocks, blockSize >> > (sets, sets0, sets1, sets2, filter, layer, pureCell);
            hipDeviceSynchronize();
            subrounds++;

            if (*pureCell == true) {
                *layer = *layer + 1;
                *pureCell = false;
            }

            // countPureCells(filter, sets2, 2);
            peel_set2 << <numBlocks, blockSize >> > (sets, sets0, sets1, sets2, filter, layer, pureCell);
            hipDeviceSynchronize();
            subrounds++;

            if (*pureCell == true) {
                *layer = *layer + 1;
                *pureCell = false;
            }

        }
        
        peel_t = clock() - peel_t;
        time_taken = ((double)peel_t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to peel %u values. \n", time_taken, size);

        clock_t recover_t = clock();
        hipFree(pureCell);
        hipFree(sets0);
        hipFree(sets1);
        hipFree(sets2);
        
        size_t recover_cnt = 0;
        recover_cnt = thrust::count_if(thrust::device, sets, sets + arrayLength, is_recovered());
        hipDeviceSynchronize();

        recover_t = clock() - recover_t;
        time_taken = ((double)recover_t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to free memory and check recovery over %zu values. \n", time_taken, arrayLength);

        if (recover_cnt == size) {
            // success
            break;
        }

        printf("peel failure, recover_cnt=%zu\n", recover_cnt);
        fflush(stdout);
        filter->seed = fuse_rng_splitmix64(&rng_counter);
    }
    
    clock_t sort_t = clock();
    uint32_t* layer_counts = sortcount_layers(sets, arrayLength, *layer);
    sort_t = clock() - sort_t;
    double time_sort = ((double)sort_t) / CLOCKS_PER_SEC; // seconds
    printf("It took %f to sort and count %zu elements\n", time_sort, arrayLength);
   
    clock_t assign_t = clock();
    size_t layer_size = sets[0].layer;
    size_t offset = 0;

    for (size_t i = 0; i < layer_size; i++) {
        assign_compacted << <numBlocks, blockSize >> > (filter, sets + offset, i, layer_counts[i]);
        hipDeviceSynchronize();
        offset += layer_counts[i];
    }

    hipFree(layer_counts);
    
    assign_t = clock() - assign_t;
    double time_taken = ((double) assign_t) / CLOCKS_PER_SEC; // in seconds
    printf("It took %f seconds to assign over %u values, %zu layers. \n", time_taken, size, *layer);
    
    hipFree(layer);
    hipFree(sets);

    return true;
}

static inline bool fuse8_contain(uint64_t key, const fuse8_t* filter) {
    uint64_t hash = fuse_mix_split(key, filter->seed);
    uint8_t f = fuse_fingerprint(hash);
    uint32_t r0 = (uint32_t)hash;
    uint32_t r1 = (uint32_t)fuse_rotl64(hash, 21);
    uint32_t r2 = (uint32_t)fuse_rotl64(hash, 42);
    uint32_t r3 = (0xBF58476D1CE4E5B9 * hash) >> 32;
    uint32_t seg = fuse_reduce(r0, FUSE_SEGMENT_COUNT);
    uint32_t h0 = (seg + 0) * filter->segmentLength + fuse_reduce(r1, filter->segmentLength);
    uint32_t h1 = (seg + 1) * filter->segmentLength + fuse_reduce(r2, filter->segmentLength);
    uint32_t h2 = (seg + 2) * filter->segmentLength + fuse_reduce(r3, filter->segmentLength);
    return f == (filter->fingerprints[h0] ^ filter->fingerprints[h1] ^
        filter->fingerprints[h2]);
}

// release memory
static inline void fuse8_free(fuse8_t* filter) {
    hipFree(filter->fingerprints);
    filter->fingerprints = NULL;
    filter->segmentLength = 0;
    hipFree(filter);
}

bool testfuse8() {
    printf("testing fuse8\n");

    fuse8_t* filter;
    hipMallocManaged(&filter, sizeof(fuse8_t));

    size_t size = 1000000;
    fuse8_allocate(size, filter);
    // we need some set of values
    uint64_t* big_set;
    hipMallocManaged(&big_set, size * sizeof(uint64_t));

    for (size_t i = 0; i < size; i++) {
        big_set[i] = i; // we use contiguous values
    }
    // we construct the filter
    fuse8_populate(big_set, size, filter);
    for (size_t i = 0; i < size; i++) {
        if (!fuse8_contain(big_set[i], filter)) {
            printf("bug!\n");
            return false;
        }
    }

    size_t random_matches = 0;
    size_t trials = 10000000; //(uint64_t)rand() << 32 + rand()
    for (size_t i = 0; i < trials; i++) {
        uint64_t random_key = ((uint64_t)rand() << 32) + rand();
        if (fuse8_contain(random_key, filter)) {
            if (random_key >= size) {
                random_matches++;
            }
        }
    }
    printf("fpp %3.10f (estimated) \n", random_matches * 1.0 / trials);
    printf("bits per entry %3.1f\n", fuse8_size_in_bytes(filter) * 8.0 / size);
    fuse8_free(filter);
    hipFree(big_set);
    return true;
}

bool testfuse8(size_t size) {
    printf("testing fuse8 ");
    printf("size = %zu \n", size);

    fuse8_t* filter;
    hipMallocManaged(&filter, sizeof(fuse8_t));

    fuse8_allocate(size, filter);
    // we need some set of values
    uint64_t* big_set;
    hipMallocManaged(&big_set, size * sizeof(uint64_t));

    for (size_t i = 0; i < size; i++) {
        big_set[i] = i; // we use contiguous values
    }
    // we construct the filter
    fuse8_populate(big_set, size, filter); // warm the cache
    for (size_t times = 0; times < 5; times++) {
        clock_t t;
        t = clock();
        fuse8_populate(big_set, size, filter);
        t = clock() - t;
        double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
        printf("It took %f seconds to build an index over %zu values. \n",
            time_taken, size);
    }
    fuse8_free(filter);
    hipFree(big_set);
    return true;
}

int main() {
    for (size_t s = 1000000; s <= 1000000; s += 1) {
        // unit test
        testfuse8();
        // performance test
        testfuse8(s);
        printf("\n");
    }
}